
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
 
#define N 1023
 
__global__ void scaleArray(float* array, float value) {
  int threadGlobalID    = threadIdx.x + blockIdx.x * blockDim.x;
  array[threadGlobalID] = array[threadGlobalID]*value;
  return;
}
 
int main() {
  float* array;
  hipMallocManaged(&array, N*sizeof(float)); // Allocate, visible to both CPU and GPU
  for (int i=0; i<N; i++) array[i] = 1.0f;    // Initialize array
 
  printf("Before: Array 0, 1 .. N-1: %f %f %f\n", array[0], array[1], array[N-1]);
  scaleArray<<<4, 256>>>(array, 3.0);
  hipDeviceSynchronize();
 
  printf("After : Array 0, 1 .. N-1: %f %f %f\n", array[0], array[1], array[N-1]);
  assert(array[N/2] == 3.0); // Check it's worked
  exit(0);
}
